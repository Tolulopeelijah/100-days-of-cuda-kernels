#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel to perform vector addition
__global__ void vecAdd(float *d_A, float *d_B, float *d_C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Compute global index
    if (i < n) {
        d_C[i] = d_A[i] + d_B[i];  // Perform addition
    }
}

// Host function
int main() {
    int n = 1000000;  // Number of elements in the vectors
    size_t size = n * sizeof(float);

    // Allocate memory on the host
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);
    
    // Initialize input vectors with some values
    for (int i = 0; i < n; i++) {
        h_A[i] = i * 0.5f;
        h_B[i] = i * 2.0f;
    }

    // Allocate memory on the device
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define CUDA kernel launch parameters
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize; // Ensure we cover all elements

    // Launch CUDA kernel
    vecAdd<<<gridSize, blockSize>>>(d_A, d_B, d_C, n);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify correctness (optional)
    int correct = 1;
    for (int i = 0; i < n; i++) {
        if (h_C[i] != (h_A[i] + h_B[i])) {
            correct = 0;
            break;
        }
    }
    
    if (correct)
        printf("Vector addition completed successfully!\n");
    else
        printf("Error: Incorrect results detected.\n");

    // Free allocated memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
#include <stdlib.h>


void add(int *d_a, int *d_b, int *d_c, int n){
     for (int i = 0; i < n; i++){
     	 d_c[i] = d_a[i] + d_b[i];
	 }
	 }

int main(){
    int n; // size of the array
    int d_a = malloc(n * sizeof(int));
    #include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel to perform vector addition
__global__ void vecAdd(float *d_A, float *d_B, float *d_C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Compute global index
    if (i < n) {
        d_C[i] = d_A[i] + d_B[i];  // Perform addition
    }
}

// Host function
int main() {
    int n = 1000000;  // Number of elements in the vectors
    size_t size = n * sizeof(float);

    // Allocate memory on the host
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);
    
    // Initialize input vectors with some values
    for (int i = 0; i < n; i++) {
        h_A[i] = i * 0.5f;
        h_B[i] = i * 2.0f;
    }

    // Allocate memory on the device
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define CUDA kernel launch parameters
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize; // Ensure we cover all elements

    // Launch CUDA kernel
    vecAdd<<<gridSize, blockSize>>>(d_A, d_B, d_C, n);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify correctness (optional)
    int correct = 1;
    for (int i = 0; i < n; i++) {
        if (h_C[i] != (h_A[i] + h_B[i])) {
            correct = 0;
            break;
        }
    }
    
    if (correct)
        printf("Vector addition completed successfully!\n");
    else
        printf("Error: Incorrect results detected.\n");

    // Free allocated memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
